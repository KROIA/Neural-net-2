#include "hip/hip_runtime.h"
﻿#include "../inc/net_kernel.cuh"

#define BLOCK_DIM 16

namespace NeuronalNet 
{
    __global__ void transpose(float* odata, float* idata, int width, int height)
    {
        __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

        // read the matrix tile into shared memory
            // load one element per thread from device memory (idata) and store it
            // in transposed order in block[][]
        unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
        unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
        if ((xIndex < width) && (yIndex < height))
        {
            unsigned int index_in = yIndex * width + xIndex;
            block[threadIdx.y][threadIdx.x] = idata[index_in];
        }

        // synchronise to ensure all writes to block[][] have completed
        __syncthreads();

        // write the transposed matrix tile to global memory (odata) in linear order
        xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
        yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
        if ((xIndex < height) && (yIndex < width))
        {
            unsigned int index_out = yIndex * height + xIndex;
            odata[index_out] = block[threadIdx.x][threadIdx.y];
        }
    }
    __host__ 
        double GPU_CUDA_transposeMatrix(float* d_list, size_t width)
    {/*
        size_t maxElement = gaussSum(width);
        size_t elementCount = width * width;

        float* d_list1;
        float* d_list2;
        GPU_CUDA_allocMem(d_list1, elementCount * sizeof(float));
        GPU_CUDA_allocMem(d_list2, elementCount * sizeof(float));
        GPU_CUDA_transferToDevice(d_list1, h_list, elementCount * sizeof(float));

        dim3 grid(width / BLOCK_DIM, width / BLOCK_DIM, 1);
        dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
        int blockSize = GPU_CUDA_getSpecs().maxThreadsPerBlock;
        int numBlocks = (maxElement - 1) / blockSize + 1;
        auto t1 = std::chrono::high_resolution_clock::now();
        transpose <<<grid, threads >>> (d_list2,d_list1, width, width);
        hipDeviceSynchronize();
        auto t2 = std::chrono::high_resolution_clock::now();
        double transposeTimeNs = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count();
        double transposeTimeMs = transposeTimeNs / 1000000;
        std::cout << "Transposetime: " << transposeTimeNs << " ns = " << transposeTimeMs << " ms\n";
        GPU_CUDA_transferToHost(d_list2, h_list, elementCount * sizeof(float));
        GPU_CUDA_freeMem(d_list1);
        GPU_CUDA_freeMem(d_list2);
        return transposeTimeMs;
        */
        size_t maxElement = gaussSum(width);
      //  size_t elementCount = width * width;

       // float* d_list;
       // GPU_CUDA_allocMem(d_list, elementCount * sizeof(float));
       // GPU_CUDA_transferToDevice(d_list, h_list, elementCount * sizeof(float));


        int blockSize = GPU_CUDA_getSpecs().maxThreadsPerBlock;
        int numBlocks = (maxElement - 1) / blockSize + 1;
        size_t sliceSize = 2048 / 4;
        if (numBlocks > sliceSize)
        {
            //std::cout << "numBlocks > 2048\n";
            numBlocks = sliceSize;
        }
        auto t1 = std::chrono::high_resolution_clock::now();
        for (size_t i = 0; i < width / sliceSize + 1; ++i)
        {
            kernel_transposeMatrix << <numBlocks, blockSize >> > (d_list, width, maxElement, gaussSum(i * sliceSize));
            
        }
        //kernel_transposeMatrix << <numBlocks, blockSize >> > (d_list, width, maxElement, 0);
        hipDeviceSynchronize();
        auto t2 = std::chrono::high_resolution_clock::now();
        double transposeTimeNs = std::chrono::duration_cast<std::chrono::nanoseconds>(t2-t1).count();
        double transposeTimeMs = transposeTimeNs / 1000000;
        //std::cout << "Transposetime: " << transposeTimeNs <<" ns = " << transposeTimeMs << " ms\n";
      //  GPU_CUDA_transferToHost(d_list, h_list, elementCount * sizeof(float));
      //  GPU_CUDA_freeMem(d_list);
        return transposeTimeMs;
    }
    __host__
        double GPU_CUDA_transposeMatrix2(float* d_list1, float* d_list2, size_t width)
    {
        size_t maxElement = gaussSum(width);
        size_t elementCount = width * width;

      // float* d_list1;
      // float* d_list2;
      // GPU_CUDA_allocMem(d_list1, elementCount * sizeof(float));
      // GPU_CUDA_allocMem(d_list2, elementCount * sizeof(float));
      // GPU_CUDA_transferToDevice(d_list1, h_list, elementCount * sizeof(float));

        dim3 grid(width / BLOCK_DIM, width / BLOCK_DIM, 1);
        dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
        int blockSize = GPU_CUDA_getSpecs().maxThreadsPerBlock;
        int numBlocks = (maxElement - 1) / blockSize + 1;
        auto t1 = std::chrono::high_resolution_clock::now();
        transpose <<<grid, threads >>> (d_list2,d_list1, width, width);
        hipDeviceSynchronize();
        auto t2 = std::chrono::high_resolution_clock::now();
        double transposeTimeNs = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count();
        double transposeTimeMs = transposeTimeNs / 1000000;
        //std::cout << "Transposetime: " << transposeTimeNs << " ns = " << transposeTimeMs << " ms\n";
      //  GPU_CUDA_transferToHost(d_list2, h_list, elementCount * sizeof(float));
      //  GPU_CUDA_freeMem(d_list1);
      //  GPU_CUDA_freeMem(d_list2);
        return transposeTimeMs;
    }


    __host__ 
        hipDeviceProp_t GPU_CUDA_getSpecs()
    {
        hipDeviceProp_t h_deviceProp;
        hipGetDeviceProperties(&h_deviceProp, 0);
        return h_deviceProp;
    }
    __host__ 
        void GPU_CUDA_calculateNet(float* weights, float* signals, float* outpuSignals,
                               size_t inputCount, size_t hiddenX, size_t hiddenY, size_t outputCount, Activation activation)
    {
        nvtxRangePush(__FUNCTION__);
        nvtxMark("Waiting...");
        kernel_calculateNet << <1, 1 >> > (weights, signals, outpuSignals,
                                           inputCount, hiddenX, hiddenY, outputCount, activation);
        hipDeviceSynchronize();
        nvtxRangePop();
    }
    __host__
        void GPU_CUDA_getRandomWeight(float min, float max, float* h_list, size_t elements)
    {
        /*hiprandStatus* d_state;
        size_t maxThreadPerBlock = 1024;
        size_t blockSize = maxThreadPerBlock;
        size_t numBlocks = (elements - 1) / blockSize + 1;
        float* d_list;
        hipMalloc(&d_state, blockSize * numBlocks);
        hipMalloc(&d_list, elements * sizeof(float));
        kernel_randomInit <<<numBlocks, blockSize >>> (d_state);
        kernel_getRandomWeight <<<numBlocks, blockSize >>> (min,max,d_state, d_list, elements);
        hipFree(d_state);
        hipFree(d_list);*/


        size_t maxThreadPerBlock = 1024;
        size_t blockSize = maxThreadPerBlock;
        size_t numBlocks = (elements - 1) / blockSize + 1;
        hiprandGenerator_t gen;
        float* d_list;
        hipMalloc(&d_list, elements * sizeof(float));
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, d_list, elements);
        hipDeviceSynchronize();
        //kernel_scaleRandomWeight <<<numBlocks, blockSize >>> (min, max, d_list, elements);
        //hipDeviceSynchronize();
        hipMemcpy(h_list, d_list, elements * sizeof(float), hipMemcpyDeviceToHost);
       /* for (size_t i = 0; i < 100; ++i)
        {
            printf("%f\n", h_list[i]);
        }*/
        hipFree(d_list);
    }


    __host__
        void GPU_CUDA_allocMem(float*& d_list, size_t byteCount)
    {
        d_list = nullptr;
        cuda_handleError(hipMalloc(&d_list, byteCount));
    }

    __host__ 
        void GPU_CUDA_freeMem(float*& d_list)
    {
        if (!d_list)
            return;
        hipError_t err = hipFree(d_list);
        cuda_handleError(err);
        if(err == hipError_t::hipSuccess)
            d_list = nullptr;
    }

    __host__
        void GPU_CUDA_transferToDevice(float* d_list, float* h_list, size_t byteCount)
    {
        cuda_handleError(hipMemcpy(d_list, h_list, byteCount, hipMemcpyHostToDevice));
    }
    __host__
        void GPU_CUDA_transferToHost(float* d_list, float* h_list, size_t byteCount)
    {
       cuda_handleError(hipMemcpy(h_list, d_list, byteCount, hipMemcpyDeviceToHost));
    }
    __host__ 
        void GPU_CUDA_convertWeightToGPUWeight(float* d_list, size_t inputCount, size_t hiddenX, size_t hiddenY, size_t outputCount)
    {

    }


    __device__
        inline float kernel_net_activation_linear(float x)
    {
        return NET_ACTIVATION_LINEAR(x);
    }
    __device__
        inline float kernel_net_activation_gaussian(float x)
    {
        return NET_ACTIVATION_GAUSSIAN(x);
    }
    __device__
        inline float kernel_net_activation_sigmoid(float x)
    {
        return NET_ACTIVATION_SIGMOID(x);
    }

    __device__ 
    kernel_ActFp* kernel_net_getActivationFunction(Activation act)
    {
        switch (act)
        {
            default:
            case Activation::sigmoid:
                return &kernel_net_activation_sigmoid;
            case Activation::linear:
                return &kernel_net_activation_linear;
            case Activation::gauss:
                return &kernel_net_activation_gaussian;

        }
    }

    __global__
        void kernel_net_calculateLayer(float* weights, float* inputSignals, float* outputSignals,
                                          size_t neuronCount, size_t inputSignalCount, kernel_ActFp* act)
    {
        size_t index = blockIdx.x * blockDim.x + threadIdx.x;
        
        

        
        if (neuronCount == 0 || index >= neuronCount)
            return;

      
        

        float res = 0;
        //weights += index * inputSignalCount;

       // ramStorage = (Storage*)weights;
       // storage = *ramStorage;
       
       
       
       /*
        for (size_t i = 0; i < inputSignalCount; ++i)
        {
            res += weights[i] * inputSignals[i];
        }*/

        const short tileSize = 32;
        __shared__ float sharedSignals[tileSize];
        
        size_t signalsBegin = 0;
        size_t signalsEnd   = 0;
        short tiles = inputSignalCount / tileSize + 1;
        short loadCount = tileSize / neuronCount + 1;
        short loadIndex = loadCount * threadIdx.x;


        for (short tile = 0; tile <tiles; ++tile)
        {
            signalsBegin = signalsEnd;
            if (tile == tiles - 1)
                signalsEnd = inputSignalCount;
            else
                signalsEnd = (tile + 1) * tileSize;


            
            for (short i = 0; i < loadCount; ++i)
            {
                short signalIndex = i + loadIndex;

                if (signalsEnd > (signalIndex + signalsBegin))
                {
                     sharedSignals[signalIndex] = inputSignals[signalIndex+signalsBegin];
                }
                __syncthreads();
            }


            for (size_t i = signalsBegin; i < signalsEnd; ++i)
            {
                //float weight = weights[index * (i+1)];


               // res += *((float*)(&storage) + storageCount) * sharedSignals[i - signalsBegin];
                
                
                float weight = weights[index + inputSignalCount * i];
                //float weight = weights[index * inputSignalCount + i];
                __syncthreads();
                res += weight * sharedSignals[i - signalsBegin];
                
                //res += sharedSignals[i - signalsBegin];

            //   if ((++storageCount) >= 1)
            //   {
            //       storageCount = 0;
            //
            //       if ((i + 8) < signalsEnd)
            //       {
            //
            //           storage = *ramStorage;
            //           ++ramStorage;
            //       }
            //       else
            //       {
            //           /*for (size_t j = i; j < signalsEnd; ++j)
            //           {
            //               *((float*)&storage + j) = weights[j];
            //           }*/
            //       }
            //       
            //   }
            }
        }

        //__syncthreads();
        outputSignals[index] = (*act)(res);
    }

    __global__
        void kernel_calculateNet(float* weights, float* signals, float* outpuSignals,
                        size_t inputCount, size_t hiddenX, size_t hiddenY, size_t outputCount, Activation act)
    {
        
        kernel_ActFp* actPtr = kernel_net_getActivationFunction(act);
        
        size_t maxThreadPerBlock = 1024;

        size_t blockSize = maxThreadPerBlock;
        size_t numBlocks = (hiddenY - 1) / blockSize + 1;
        float* tmpHiddenOutSignals1 = new float[hiddenY];
        float* tmpHiddenOutSignals2 = new float[hiddenY];
        hipDeviceSynchronize();
        //kernel_net_calculateLayer <<< numBlocks, blockSize >>> (weights, signals, tmpHiddenOutSignals1, hiddenY, inputCount, actPtr);
        kernel_net_calculateLayer <<< 1, 1 >>> (weights, signals, tmpHiddenOutSignals1, hiddenY, inputCount, actPtr);
        weights += inputCount * hiddenY;
        hipDeviceSynchronize();

       /* for (size_t i = 1; i < hiddenY; i += 100)
        {
            printf("0 %i %f\n", i, tmpHiddenOutSignals1[i]);
        }*/

        for (size_t i = 1; i < hiddenX; ++i)
        {
            kernel_net_calculateLayer <<< numBlocks, blockSize >>> (weights, tmpHiddenOutSignals1, tmpHiddenOutSignals2, hiddenY, hiddenY, actPtr);
            weights += hiddenY * hiddenY;
            float* tmp = tmpHiddenOutSignals1;
            tmpHiddenOutSignals1 = tmpHiddenOutSignals2;
            tmpHiddenOutSignals2 = tmp;
            hipDeviceSynchronize();
            /*for (size_t j = 0; j < hiddenY; j += 100)
            {
                float ttt = tmpHiddenOutSignals2[j];
                printf("%i %i %f\n", i,j, ttt);
            }*/
            //printf("v2 %f\n", v2);
            
        }

        numBlocks = (outputCount - 1) / blockSize + 1;
        kernel_net_calculateLayer <<< numBlocks, blockSize >>> (weights, tmpHiddenOutSignals1, outpuSignals, outputCount, hiddenY, actPtr);
        hipDeviceSynchronize();
       /* for (size_t i = 0; i < outputCount; ++i)
        {
            printf("out %i %f\n", i, outpuSignals[i]);
        }*/
        delete[] tmpHiddenOutSignals1;
        delete[] tmpHiddenOutSignals2;

    }

    __global__ 
        void kernel_convertLayerWeightToGPUWeight(float* d_list, size_t signalCount, size_t neuronCount)
    {
        

        /*
        1) Kreisläufe finden:
            index 0 überspringen
            index 1 index speichern, kreislauf durchrechnen bis zum anfang, index in Liste aufnehmen
            index++ -> wieder index speichern, kreislauf druchrechnen bis anfang. bei jedem element prüfen, 
                       ob der jeweilige index schon in der Liste aufgenommen ist, wenn ja ist der Kreislauf
                       schon notiert.

            index++ -> wiederholen

            --> Ergiebt eine Liste von startindexen für die kreisläufe.

            --> 
        
        */

        size_t weightCount = signalCount * neuronCount;
        size_t circuitStartIndex[2];
        circuitStartIndex[0] = 0;
        circuitStartIndex[1] = 0;
        size_t circuitIndex = 0;

        // Kreisläufe finden
        // Erstes und letztes Elemnt überspringen, diese bleiben konstant.
        for (size_t currentStartIndex = 1; currentStartIndex < weightCount-1; ++currentStartIndex)
        {
            size_t destinationIndex = 0;
            size_t srcIndex = currentStartIndex;
            bool circuitAlreadyExists = false;
            while (destinationIndex != currentStartIndex) 
            {
                kernel_convertLayerWeightToGPUWeight_getNewIndex(srcIndex, destinationIndex, signalCount, neuronCount);
                
                for (int i = 0; i < 2; ++i)
                {
                    if (circuitStartIndex[i] == destinationIndex)
                    {
                        circuitAlreadyExists = true;
                    }
                }
                srcIndex = destinationIndex;
            }      
            if (!circuitAlreadyExists)
            {
                circuitStartIndex[circuitIndex] = currentStartIndex;
                ++circuitIndex;
                if (circuitIndex >= 2)
                {
                    printf("ERROR: more than 2 circuits found\n");
                }
            }
        }

/*
        size_t inputIndex = ;
        size_t outputIndex = inputIndex / neuronCount + (inputIndex % neuronCount) * signalCount;
        result[inputIndex / h][inputIndex % h] = m[x][y];*/

        /*
        size_t weightCount = signalCount * neuronCount -1;
        float tmp;
        size_t destIndex = 0;
        size_t srcIndex =  1;

        size_t srcIndexX = 1;
        size_t srcIncexY = 0;
        size_t destIndexX = 0;
        size_t destIncexY = 0;
        for (size_t x = 0; x <signalCount; ++x)
        {
            for (size_t y = 0; y < neuronCount; ++y)
            {
                if (x + y == 0 || x + y == signalCount + neuronCount-2)
                    continue;
                
               // destIndexX = (srcIndexX * signalCount) % signalCount;
               // destIndexY = (srcIndexY *)




                tmp = d_list[destIndex];
                d_list[destIndex] = d_list[srcIndex];
                destIndex = (destIndex + signalCount) % weightCount;
                ++srcIndex;
            }
        }*/
    }

    __global__ 
        void kernel_transposeMatrix(float* d_list, size_t width, size_t maxIndex, size_t indexOffset)
    {
        size_t index = blockIdx.x * blockDim.x + threadIdx.x + indexOffset;
       // #define COALESCED
        //maxIndex = kernel_gaussSum(width)
      
        size_t x = kernel_invGaussSum(index);
        size_t y = index - kernel_gaussSum(x);

        if (index >= maxIndex || x==y)
        {
            //printf(" returning index: %i\tx: %i\ty: %i\n", index, x, y);
            return;
        }
        if (y != 0)
            return;
       // if (index == 1)
        //    printf("maxIndex: %i\n", maxIndex);
        //if(x==0)
        //printf(" i: %3i x: %3i\n", index, x);

        if (x > width || y > width)
            printf("ERROR: x>width || y>width\n");
        size_t elementIndex1 = y * width + x;
        size_t elementIndex2 = x * width + y;
       // if (index == 2098176)
       //     printf("index == 2098176, d_list[%i][%i] = %1.1f\n", x, y, d_list[elementIndex1]);

       // if (elementIndex1 == 1 || elementIndex2 == 32)
      //      printf("");


#ifdef COALESCED
        //__shared__ float buffer1[1024];
        __shared__ float buffer2[1024];

        
        //buffer1[threadIdx.x] = d_list[elementIndex1];
        //__syncthreads();
        buffer2[threadIdx.x] = d_list[elementIndex2];
        __syncthreads();
        d_list[elementIndex1] = buffer2[threadIdx.x];
        __syncthreads();
        d_list[elementIndex2] = d_list[elementIndex1];

#else
        
        float tmp = d_list[elementIndex1];
        d_list[elementIndex1] = d_list[elementIndex2];
        d_list[elementIndex2] = tmp;
#endif
    }


    __host__ 
        size_t gaussSum(size_t val)
    {
        return (val * (val + 1)) / 2;
    }
    
    __device__ 
        size_t kernel_gaussSum(size_t val)
    {
        return (val * (val + 1)) / 2;
    }

    __host__ 
        size_t invGaussSum(size_t sum)
    {
        return (size_t)floor((sqrt(8 * (float)sum + 1) - 1) / 2);
    }
    __device__ 
        size_t kernel_invGaussSum(size_t sum)
    {
        return (size_t)floorf((sqrtf(8 * (float)sum + 1) - 1) / 2);
    }


    __device__ 
        void kernel_convertLayerWeightToGPUWeight_getNewIndex(size_t startIndex, size_t& endIndex, size_t signalCount, size_t neuronCount)
    {
        // Calculates the new Position of a element
        endIndex = startIndex / neuronCount + (startIndex % neuronCount) * signalCount;
    }

}

